#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "fframe.h"

#define LIMIT_NUM_SIGNALS 8192

#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
if (code != hipSuccess)
{
fprintf(stderr, "CUDA Error: %s at %s:%d\n",
hipGetErrorString(code), file, line);
if (abort) exit(code);
}
}
#else
#define cudaCheckError(ans) ans
#endif

struct GlobalConstants {
    int numCircuitSignals;
    int numCircuitInputs;
    int numCircuitOutputs;
    int numTestVectors;
};

__constant__ GlobalConstants cuConstParams;

__global__ void
faultSim_kernel(CudaGate* aCudaCircuitStructure, int* aCudaCircuitTraversalOrder, int* aCudaCircuitInputs,  int* aCudaCircuitOutputs,  uint8_t* aTestVectors, uint8_t* aDetectedFaults) {

    int myTestVectorIdx = blockIdx.x;
    int myThreadIdx = threadIdx.x;

    __shared__ uint8_t myCorrectOutputs[LIMIT_NUM_SIGNALS];
    uint8_t myLocalCircuitState[LIMIT_NUM_SIGNALS];

    for (int myFaultIdx = myThreadIdx; myFaultIdx < (cuConstParams.numCircuitSignals * 2) + 1; myFaultIdx += blockDim.x) {

        int myCurrTraversalIdx;

        for (myCurrTraversalIdx = 0; myCurrTraversalIdx < cuConstParams.numCircuitInputs; myCurrTraversalIdx++) {
            uint8_t myNewCircuitVal = aTestVectors[myTestVectorIdx*cuConstParams.numCircuitInputs + myCurrTraversalIdx];

            if ((myFaultIdx != 0) && ((myFaultIdx-1) / 2) == aCudaCircuitTraversalOrder[myCurrTraversalIdx]){
                if ((myFaultIdx-1) % 2 == 0) {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 0;
                } else {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 1;
                }
            } else {
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = myNewCircuitVal;
            }
        }

        for (; myCurrTraversalIdx < cuConstParams.numCircuitSignals; myCurrTraversalIdx++) {

            int myCurrentGateIdx = aCudaCircuitTraversalOrder[myCurrTraversalIdx];
            CudaGate myCurrGate = aCudaCircuitStructure[myCurrentGateIdx];
            uint8_t myNewCircuitVal = myLocalCircuitState[myCurrGate.fanin[0]];

            switch (myCurrGate.gateType)
            {
            case CudaGateType::AND:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal &= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                break;
            case CudaGateType::OR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal |= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                break;
            case CudaGateType::NOT:
                myNewCircuitVal = !myCurrGate.fanin[0];
                break;
            case CudaGateType::XOR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal ^= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                break;
            case CudaGateType::NAND:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal &= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                myNewCircuitVal = !myNewCircuitVal;
                break;
            case CudaGateType::NOR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal |= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                myNewCircuitVal = !myNewCircuitVal;
                break;
            case CudaGateType::XNOR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal ^= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                myNewCircuitVal = !myNewCircuitVal;
                break;
            case CudaGateType::BUFF: // LEAVE EMPTY
                /* code */
                break;
            case CudaGateType::INPUT: // LEAVE EMPTY
                /* code */
                printf("Error: Should not see input at this phase of traversal: %d\n", myCurrTraversalIdx);
                break;
            default:
                break;
            }

            if ((myFaultIdx != 0) && ((myFaultIdx-1) / 2) == aCudaCircuitTraversalOrder[myCurrTraversalIdx]){
                if ((myFaultIdx-1) % 2 == 0) {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 0;
                } else {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 1;
                }
            } else {
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = myNewCircuitVal;
            }
        }

        if (myFaultIdx == 0) {
            for (int i = 0; i < cuConstParams.numCircuitOutputs; i++) {

                int myOutputIdx = aCudaCircuitOutputs[i];
                myCorrectOutputs[i] = myLocalCircuitState[myOutputIdx];
            }
        }

        __syncthreads();

        if (myFaultIdx != 0){
            int myDetectedFaultsIdx = myTestVectorIdx * (cuConstParams.numCircuitSignals * 2) + myFaultIdx - 1;
            aDetectedFaults[myDetectedFaultsIdx] = 0;
            for (int i = 0; i < cuConstParams.numCircuitOutputs; i++) {

                int myOutputIdx = aCudaCircuitOutputs[i];
                if (myCorrectOutputs[i] != myLocalCircuitState[myOutputIdx]) {
                    aDetectedFaults[myDetectedFaultsIdx] = 1;
                }

            }
        }
    }
}


void cudaFaultSim(int aNumCircuitSignals, CudaGate* aCircuitStructure, int* aCircuitTraversalOrder, int aNumCircuitInputs, int* aCircuitInputs, int aNumCircuitOutputs, int* aCircuitOutputs, int aNumTestVectors, uint8_t* aTestVectors, uint8_t* aDetectedFaults) {

    // Compute number of blocks and threads per block
    const int myThreadsPerBlock = std::min((aNumCircuitSignals * 2) + 1, 768);
    const int myNumBlocks = aNumTestVectors;

    if (aNumCircuitSignals >= LIMIT_NUM_SIGNALS){
        printf("Error: Too many signals within circuit - need to increase LIMIT_NUM_SIGNALS\n");
    }

    // Allocate buffers on GPU
    CudaGate* myCudaCircuitStructure;
    int* myCudaCircuitTraversalOrder;
    int* myCudaCircuitInputs;
    int* myCudaCircuitOutputs;
    uint8_t* myCudaTestVectors;
    uint8_t* myCudaDetectedFaults;
    hipMalloc(&myCudaCircuitStructure, sizeof(CudaGate) * aNumCircuitSignals);
    hipMalloc(&myCudaCircuitTraversalOrder, sizeof(int) * aNumCircuitSignals);
    hipMalloc(&myCudaCircuitInputs, sizeof(int) * aNumCircuitInputs);
    hipMalloc(&myCudaCircuitOutputs, sizeof(int) * aNumCircuitOutputs);
    hipMalloc(&myCudaTestVectors, sizeof(uint8_t) * aNumCircuitInputs * aNumTestVectors);
    hipMalloc(&myCudaDetectedFaults, sizeof(uint8_t) * aNumCircuitSignals * 2 * aNumTestVectors);

    hipMemcpy(myCudaCircuitStructure, aCircuitStructure, sizeof(CudaGate) * aNumCircuitSignals, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitTraversalOrder, aCircuitTraversalOrder, sizeof(int) * aNumCircuitSignals, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitInputs, aCircuitInputs, sizeof(int) * aNumCircuitInputs, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitOutputs, aCircuitOutputs, sizeof(int) * aNumCircuitOutputs, hipMemcpyHostToDevice);
    hipMemcpy(myCudaTestVectors, aTestVectors, sizeof(uint8_t) * aNumCircuitInputs * aNumTestVectors, hipMemcpyHostToDevice);

    GlobalConstants params;
    params.numCircuitSignals = aNumCircuitSignals;
    params.numCircuitInputs = aNumCircuitInputs;
    params.numCircuitOutputs = aNumCircuitOutputs;
    params.numTestVectors = aNumTestVectors;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

    // Run kernel
    faultSim_kernel<<<myNumBlocks, myThreadsPerBlock>>>(myCudaCircuitStructure, myCudaCircuitTraversalOrder, myCudaCircuitInputs, myCudaCircuitOutputs, myCudaTestVectors, myCudaDetectedFaults);
    cudaCheckError(hipDeviceSynchronize());

    hipMemcpy(aDetectedFaults, myCudaDetectedFaults, sizeof(uint8_t) * aNumCircuitSignals * 2 * aNumTestVectors, hipMemcpyDeviceToHost);
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
