#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "fframe.h"

// Data structure memory limits
#define LIMIT_NUM_SIGNALS 8192

// CUDA error checking
#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
if (code != hipSuccess)
{
fprintf(stderr, "CUDA Error: %s at %s:%d\n",
hipGetErrorString(code), file, line);
if (abort) exit(code);
}
}
#else
#define cudaCheckError(ans) ans
#endif

// Read-only global constant during kernel invocation
struct GlobalConstants {
    int numCircuitSignals;
    int numCircuitInputs;
    int numCircuitOutputs;
    int numTestVectors;
};

__constant__ GlobalConstants cuConstParams;

// Parallel implementation of fault simulation provided the CUDA-friendly input and output data structures
__global__ void
faultSim_kernel(CudaGate* aCudaCircuitStructure, int* aCudaCircuitTraversalOrder, int* aCudaCircuitInputs,  int* aCudaCircuitOutputs,  uint8_t* aTestVectors, uint8_t* aDetectedFaults) {

    int myTestVectorIdx = blockIdx.x;
    int myThreadIdx = threadIdx.x;

    // Initialize shared golden outputs array and circuit state array
    __shared__ uint8_t myCorrectOutputs[LIMIT_NUM_SIGNALS];
    uint8_t myLocalCircuitState[LIMIT_NUM_SIGNALS];

    // Thread iterates through all responsible faults
    for (int myFaultIdx = myThreadIdx; myFaultIdx < (cuConstParams.numCircuitSignals * 2) + 1; myFaultIdx += blockDim.x) {

        int myCurrTraversalIdx;

        // Traverse through the inputs of the circuit to populate the state
        for (myCurrTraversalIdx = 0; myCurrTraversalIdx < cuConstParams.numCircuitInputs; myCurrTraversalIdx++) {
            uint8_t myNewCircuitVal = aTestVectors[myTestVectorIdx*cuConstParams.numCircuitInputs + myCurrTraversalIdx];

            // Override state if the signal corresponds to the current faultIdx
            if ((myFaultIdx != 0) && ((myFaultIdx-1) / 2) == aCudaCircuitTraversalOrder[myCurrTraversalIdx]){
                if ((myFaultIdx-1) % 2 == 0) {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 0;
                } else {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 1;
                }
            } else {
                // Write correct circuit value
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = myNewCircuitVal;
            }
        }

        // Iterate through all other circuit signals after inputs
        for (; myCurrTraversalIdx < cuConstParams.numCircuitSignals; myCurrTraversalIdx++) {

            int myCurrentGateIdx = aCudaCircuitTraversalOrder[myCurrTraversalIdx];
            CudaGate myCurrGate = aCudaCircuitStructure[myCurrentGateIdx];
            uint8_t myNewCircuitVal = myLocalCircuitState[myCurrGate.fanin[0]];

            // Update circuit signal state based on signal type
            switch (myCurrGate.gateType)
            {
            // Determine output value given all current inputs (guaranteed to be determinable due to predetermined traversal order)
            case CudaGateType::AND:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal &= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                break;
            case CudaGateType::OR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal |= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                break;
            case CudaGateType::NOT:
                myNewCircuitVal = !myCurrGate.fanin[0];
                break;
            case CudaGateType::XOR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal ^= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                break;
            case CudaGateType::NAND:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal &= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                myNewCircuitVal = !myNewCircuitVal;
                break;
            case CudaGateType::NOR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal |= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                myNewCircuitVal = !myNewCircuitVal;
                break;
            case CudaGateType::XNOR:
                for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                    myNewCircuitVal ^= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
                }
                myNewCircuitVal = !myNewCircuitVal;
                break;
            case CudaGateType::BUFF: // LEAVE EMPTY
                /* code */
                break;
            case CudaGateType::INPUT: // LEAVE EMPTY
                /* code */
                printf("Error: Should not see input at this phase of traversal: %d\n", myCurrTraversalIdx);
                break;
            default:
                break;
            }

            // Override state if the signal corresponds to the current faultIdx
            if ((myFaultIdx != 0) && ((myFaultIdx-1) / 2) == aCudaCircuitTraversalOrder[myCurrTraversalIdx]){
                if ((myFaultIdx-1) % 2 == 0) {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 0;
                } else {
                    myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 1;
                }
            } else {
                // Write correct circuit value
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = myNewCircuitVal;
            }
        }

        // Write shared golden outputs if myFaultIdx == 0
        if (myFaultIdx == 0) {
            for (int i = 0; i < cuConstParams.numCircuitOutputs; i++) {

                int myOutputIdx = aCudaCircuitOutputs[i];
                myCorrectOutputs[i] = myLocalCircuitState[myOutputIdx];
            }
        }

        __syncthreads();

        // All other faultIdx/threadIdx compare local output states against shared golden states and populates global detection data structure
        if (myFaultIdx != 0){
            int myDetectedFaultsIdx = myTestVectorIdx * (cuConstParams.numCircuitSignals * 2) + myFaultIdx - 1;
            aDetectedFaults[myDetectedFaultsIdx] = 0;
            for (int i = 0; i < cuConstParams.numCircuitOutputs; i++) {

                int myOutputIdx = aCudaCircuitOutputs[i];
                if (myCorrectOutputs[i] != myLocalCircuitState[myOutputIdx]) {
                    aDetectedFaults[myDetectedFaultsIdx] = 1;
                }

            }
        }
    }
}


// CUDA entry point - initialize memory and invoke kernel
void cudaFaultSim(int aNumCircuitSignals, CudaGate* aCircuitStructure, int* aCircuitTraversalOrder, int aNumCircuitInputs, int* aCircuitInputs, int aNumCircuitOutputs, int* aCircuitOutputs, int aNumTestVectors, uint8_t* aTestVectors, uint8_t* aDetectedFaults) {

    // Compute number of blocks and threads per block
    const int myThreadsPerBlock = std::min((aNumCircuitSignals * 2) + 1, 768);
    const int myNumBlocks = aNumTestVectors;

    if (aNumCircuitSignals >= LIMIT_NUM_SIGNALS){
        printf("Error: Too many signals within circuit - need to increase LIMIT_NUM_SIGNALS\n");
    }

    // Allocate buffers on GPU
    CudaGate* myCudaCircuitStructure;
    int* myCudaCircuitTraversalOrder;
    int* myCudaCircuitInputs;
    int* myCudaCircuitOutputs;
    uint8_t* myCudaTestVectors;
    uint8_t* myCudaDetectedFaults;
    hipMalloc(&myCudaCircuitStructure, sizeof(CudaGate) * aNumCircuitSignals);
    hipMalloc(&myCudaCircuitTraversalOrder, sizeof(int) * aNumCircuitSignals);
    hipMalloc(&myCudaCircuitInputs, sizeof(int) * aNumCircuitInputs);
    hipMalloc(&myCudaCircuitOutputs, sizeof(int) * aNumCircuitOutputs);
    hipMalloc(&myCudaTestVectors, sizeof(uint8_t) * aNumCircuitInputs * aNumTestVectors);
    hipMalloc(&myCudaDetectedFaults, sizeof(uint8_t) * aNumCircuitSignals * 2 * aNumTestVectors);

    // Perform memcpy of data structures
    hipMemcpy(myCudaCircuitStructure, aCircuitStructure, sizeof(CudaGate) * aNumCircuitSignals, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitTraversalOrder, aCircuitTraversalOrder, sizeof(int) * aNumCircuitSignals, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitInputs, aCircuitInputs, sizeof(int) * aNumCircuitInputs, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitOutputs, aCircuitOutputs, sizeof(int) * aNumCircuitOutputs, hipMemcpyHostToDevice);
    hipMemcpy(myCudaTestVectors, aTestVectors, sizeof(uint8_t) * aNumCircuitInputs * aNumTestVectors, hipMemcpyHostToDevice);

    // Populate kernel global constants
    GlobalConstants params;
    params.numCircuitSignals = aNumCircuitSignals;
    params.numCircuitInputs = aNumCircuitInputs;
    params.numCircuitOutputs = aNumCircuitOutputs;
    params.numTestVectors = aNumTestVectors;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

    // Run kernel
    faultSim_kernel<<<myNumBlocks, myThreadsPerBlock>>>(myCudaCircuitStructure, myCudaCircuitTraversalOrder, myCudaCircuitInputs, myCudaCircuitOutputs, myCudaTestVectors, myCudaDetectedFaults);
    cudaCheckError(hipDeviceSynchronize());

    // Perform memcpy of results
    hipMemcpy(aDetectedFaults, myCudaDetectedFaults, sizeof(uint8_t) * aNumCircuitSignals * 2 * aNumTestVectors, hipMemcpyDeviceToHost);
}


void
printCudaInfo() {
    // for fun, just print out some stats on the machine
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
