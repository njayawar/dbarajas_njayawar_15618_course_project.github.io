#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "fframe.h"

#define LIMIT_NUM_SIGNALS 1024

struct GlobalConstants {
    int numCircuitSignals;
    int numCircuitInputs;
    int numCircuitOutputs;
    int numTestVectors;
};

__constant__ GlobalConstants cuConstParams;

__global__ void
faultSim_kernel(CudaGate* aCudaCircuitStructure, int* aCudaCircuitTraversalOrder, int* aCudaCircuitInputs,  int* aCudaCircuitOutputs,  uint8_t* aTestVectors, uint8_t* aDetectedFaults) {

    int myTestVectorIdx = blockIdx.x;
    int myFaultIdx = threadIdx.x;

    __shared__ uint8_t myCorrectOutputs[LIMIT_NUM_SIGNALS];

    uint8_t myLocalCircuitState[LIMIT_NUM_SIGNALS];
    int myCurrTraversalIdx;

    for (myCurrTraversalIdx = 0; myCurrTraversalIdx < cuConstParams.numCircuitInputs; myCurrTraversalIdx++) {
        uint8_t myNewCircuitVal = aTestVectors[myTestVectorIdx*cuConstParams.numCircuitInputs + myCurrTraversalIdx];

        if ((myFaultIdx != 0) && ((myFaultIdx-1) / 2) == aCudaCircuitTraversalOrder[myCurrTraversalIdx]){
            if ((myFaultIdx-1) % 2 == 0) {
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 0;
            } else {
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 1;
            }
        } else {
            myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = myNewCircuitVal;
        }
    }

    for (; myCurrTraversalIdx < cuConstParams.numCircuitSignals; myCurrTraversalIdx++) {

        int myCurrentGateIdx = aCudaCircuitTraversalOrder[myCurrTraversalIdx];
        CudaGate myCurrGate = aCudaCircuitStructure[myCurrentGateIdx];
        uint8_t myNewCircuitVal = myLocalCircuitState[myCurrGate.fanin[0]];

        switch (myCurrGate.gateType)
        {
        case CudaGateType::AND:
            for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                myNewCircuitVal &= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
            }
            break;
        case CudaGateType::OR:
            for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                myNewCircuitVal |= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
            }
            break;
        case CudaGateType::NOT:
            myNewCircuitVal = !myCurrGate.fanin[0];
            break;
        case CudaGateType::XOR:
            for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                myNewCircuitVal ^= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
            }
            break;
        case CudaGateType::NAND:
            for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                myNewCircuitVal &= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
            }
            myNewCircuitVal = !myNewCircuitVal;
            break;
        case CudaGateType::NOR:
            for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                myNewCircuitVal |= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
            }
            myNewCircuitVal = !myNewCircuitVal;
            break;
        case CudaGateType::XNOR:
            for (int myInputIdx = 1; myInputIdx < myCurrGate.faninSize; myInputIdx++) {
                myNewCircuitVal ^= myLocalCircuitState[myCurrGate.fanin[myInputIdx]];
            }
            myNewCircuitVal = !myNewCircuitVal;
            break;
        case CudaGateType::BUFF: // LEAVE EMPTY
            /* code */
            break;
        case CudaGateType::INPUT: // LEAVE EMPTY
            /* code */
            printf("Error: Should not see input at this phase of traversal: %d\n", myCurrTraversalIdx);
            break;
        default:
            break;
        }

        if ((myFaultIdx != 0) && ((myFaultIdx-1) / 2) == aCudaCircuitTraversalOrder[myCurrTraversalIdx]){
            if ((myFaultIdx-1) % 2 == 0) {
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 0;
            } else {
                myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = 1;
            }
        } else {
            myLocalCircuitState[aCudaCircuitTraversalOrder[myCurrTraversalIdx]] = myNewCircuitVal;
        }
    }

    __syncthreads();

    if (myFaultIdx == 0) {
        for (int i = 0; i < cuConstParams.numCircuitOutputs; i++) {

            int myOutputIdx = aCudaCircuitOutputs[i];
            myCorrectOutputs[i] = myLocalCircuitState[myOutputIdx];

        }
    }

    __syncthreads();

    if (myFaultIdx != 0){
        int myDetectedFaultsIdx = myTestVectorIdx * (cuConstParams.numCircuitSignals * 2) + myFaultIdx - 1;
        aDetectedFaults[myDetectedFaultsIdx] = 0;
        for (int i = 0; i < cuConstParams.numCircuitOutputs; i++) {

            int myOutputIdx = aCudaCircuitOutputs[i];
            if (myCorrectOutputs[i] != myLocalCircuitState[myOutputIdx]) {
                aDetectedFaults[myDetectedFaultsIdx] = 1;
                printf("Found detected fault on output %d | Good: %d | Bad: %d\n", myOutputIdx, myCorrectOutputs[i], myLocalCircuitState[myOutputIdx]);
            }

        }
    }
}


void cudaFaultSim(int aNumCircuitSignals, CudaGate* aCircuitStructure, int* aCircuitTraversalOrder, int aNumCircuitInputs, int* aCircuitInputs, int aNumCircuitOutputs, int* aCircuitOutputs, int aNumTestVectors, uint8_t* aTestVectors, uint8_t* aDetectedFaults) {

    // Compute number of blocks and threads per block
    const int myThreadsPerBlock = (aNumCircuitSignals * 2) + 1;
    const int myNumBlocks = aNumTestVectors;

    if (aNumCircuitSignals >= LIMIT_NUM_SIGNALS){
        printf("Error: Too many signals within circuit - need to increase LIMIT_NUM_SIGNALS\n");
    }

    // Allocate buffers on GPU
    CudaGate* myCudaCircuitStructure;
    int* myCudaCircuitTraversalOrder;
    int* myCudaCircuitInputs;
    int* myCudaCircuitOutputs;
    uint8_t* myCudaTestVectors;
    uint8_t* myCudaDetectedFaults;
    hipMalloc(&myCudaCircuitStructure, sizeof(CudaGate) * aNumCircuitSignals);
    hipMalloc(&myCudaCircuitTraversalOrder, sizeof(int) * aNumCircuitSignals);
    hipMalloc(&myCudaCircuitInputs, sizeof(int) * aNumCircuitInputs);
    hipMalloc(&myCudaCircuitOutputs, sizeof(int) * aNumCircuitOutputs);
    hipMalloc(&myCudaTestVectors, sizeof(uint8_t) * aNumCircuitInputs * aNumCircuitSignals);
    hipMalloc(&myCudaDetectedFaults, sizeof(uint8_t) * aNumCircuitSignals * 2 * aNumTestVectors);

    // start timing after allocation of device memory
    double startTime = CycleTimer::currentSeconds();

    hipMemcpy(myCudaCircuitStructure, aCircuitStructure, sizeof(CudaGate) * aNumCircuitSignals, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitTraversalOrder, aCircuitTraversalOrder, sizeof(int) * aNumCircuitSignals, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitInputs, aCircuitInputs, sizeof(int) * aNumCircuitInputs, hipMemcpyHostToDevice);
    hipMemcpy(myCudaCircuitOutputs, aCircuitOutputs, sizeof(int) * aNumCircuitOutputs, hipMemcpyHostToDevice);
    hipMemcpy(myCudaTestVectors, aTestVectors, sizeof(uint8_t) * aNumCircuitInputs * aNumCircuitSignals, hipMemcpyHostToDevice);

    GlobalConstants params;
    params.numCircuitSignals = aNumCircuitSignals;
    params.numCircuitInputs = aNumCircuitInputs;
    params.numCircuitOutputs = aNumCircuitOutputs;
    params.numTestVectors = aNumTestVectors;
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

    // Run kernel
    faultSim_kernel<<<myNumBlocks, myThreadsPerBlock>>>(myCudaCircuitStructure, myCudaCircuitTraversalOrder, myCudaCircuitInputs, myCudaCircuitOutputs, myCudaTestVectors, myCudaDetectedFaults);
    hipDeviceSynchronize();

    hipMemcpy(aDetectedFaults, myCudaDetectedFaults, sizeof(uint8_t) * aNumCircuitSignals * 2 * aNumTestVectors, hipMemcpyDeviceToHost);

    double endTime = CycleTimer::currentSeconds();
}

void
printCudaInfo() {

    // for fun, just print out some stats on the machine

    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
